#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <random>
#include <hiprand.h>
#include <math.h>
#include "kernels.cuh"

int main(){
    unsigned int n = 256*256;
    unsigned int m = 1<<14;
    int *h_count;
    int *d_count;
    hiprandState *d_state;
    float pi;

    //allocate memory
    h_count = (int*)malloc(sizeof(int));
    hipMalloc((int **)&d_count,sizeof(int));
    hipMalloc((hiprandState **) &d_state, n*sizeof(hiprandState));
    hipMemset(d_count,0,sizeof(int));
    //set up timing stuff
    float gpu_elapsed_time;
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);
    //set kernel 
    dim3 grid = 256;
    dim3 block = 256;
    setup_kernel<<<grid,block>>>(d_state);
    //monte carlo kernel
    monte_carlo_pi_kernel<<<grid, block,256*sizeof(int)>>>(d_state, d_count, m);
    
    //copy results back to the host
    hipMemcpy(h_count,d_count,sizeof(int),hipMemcpyDeviceToHost);
    
    hipEventRecord(gpu_stop,0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
    //display results and timings for GPU
    pi = (*h_count)*4.0/(n*m);
    std::cout<<"Approximate pi calculated on GPU is: "<<pi<<" and calculation took "<<gpu_elapsed_time<<" milli-seconds"<<std::endl;    
    
    //serial version
    clock_t cpu_start = clock();
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0,1.0);
    unsigned int count = 0;
    for(unsigned int i = 0; i < n; i++){
        int temp = 0;
        while(temp < m){
            float x = distribution(generator);
            float y = distribution(generator);
            float rr = x*x +y*y;
            if(rr <= 1){
                count++;
            }
            temp++;
        }
    }
    clock_t cpu_stop = clock();
    pi = 4.0*count/(n*m);
    std::cout<<"Approximate pi calculated on CPU is: "<<pi<<" and calculation took "<<1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC<<" milli-seconds"<<std::endl;
    //free memory
}

