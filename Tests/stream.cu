#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
 
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, size_t size);

__global__ void addKernel(int *c, const int *a){

  int i = threadIdx.x;
  extern __shared__ int smem[];
  smem[i] = a[i];
  __syncthreads();
if(i==0)//0号线程做平方和
  {
	c[0] = 0;
	for(int d = 0; d<5;d++)
	{
		c[0]+=smem[d]*smem[d];
	}
  }

if(i == 1)//1号线程做累加
  { 
	c[1] = 0;
	for(int d = 0;d<5;d++)
	{
		c[1] += smem[d];
	}
  }
 if(i == 2)//2号线程做累乘
  {
	c[2] = 1;
	for(int d = 0;d<5;d++)
	{
		c[2] *= smem[d];
	}
  }
}


int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    int c[arraySize] = { 0 };
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, arraySize);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	printf("\t1+2+3+4+5 = %d\n\t1^2+2^2+3^2+4^2+5^2 = %d\n\t1*2*3*4*5 = %d\n\n\n\n\n\n", c[1], c[0], c[2]);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}
 
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a,  size_t size)
{
    int *dev_a = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
 
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
 
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
 
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, 5, size*sizeof(int), 0>>>(dev_c, dev_a);
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
 
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
 
Error:
    hipFree(dev_c);
    hipFree(dev_a);    
    return cudaStatus;
}
