#include "hip/hip_runtime.h"
#include "./common/book.h"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel( int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N) {
        int idx1 = (idx + 1 ) % 256;
        int idx2 = (idx + 1 ) % 256;

        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void) {
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
        return 0;
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // start the timers
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    // initialize the stream
    HANDLE_ERROR( hipStreamCreate(&stream));

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c,
                              N * sizeof(int) ) );

    // allocate host locked memory, used to stream
    HANDLE_ERROR( hipHostAlloc( (void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    for(int i = 0; i < FULL_DATA_SIZE; i += N){
        // copy the locked memory to the device, async(异步copy必须是锁页内存)
        HANDLE_ERROR( hipMemcpyAsync( dev_a, host_a+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_b, host_b+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream ) );
        kernel<<<N/256,256,0,stream>>>( dev_a, dev_b, dev_c );

        // copy the data from device to locked memory
        HANDLE_ERROR( hipMemcpyAsync( host_c+i, dev_c,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream ) );
}
    //等待一个stream中的所有任务执行完成.
    HANDLE_ERROR( hipStreamSynchronize( stream ) );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );

    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );

    // cleanup the streams and memory
    HANDLE_ERROR( hipHostFree( host_a ) );
    HANDLE_ERROR( hipHostFree( host_b ) );
    HANDLE_ERROR( hipHostFree( host_c ) );
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );
    HANDLE_ERROR( hipStreamDestroy( stream ) );

    return 0;
}
