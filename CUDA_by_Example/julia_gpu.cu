#include "./common/book.h"
#include "./common/image.h"

#define DIM 1000

struct hipComplex {
    float r;
    float i;
    __device__ hipComplex( float a, float b):r(a),i(b) {}
    __device__ float magnitude2(void){ return r*r + i*i; }
 
    __device__ hipComplex operator*(const hipComplex &a){
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
     }
    __device__ hipComplex operator+(const hipComplex &a){
        return hipComplex(r+a.r, i+a.i);
    }
};



