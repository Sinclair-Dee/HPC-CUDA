#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Multi Device Cooperative Groups, also uses Unified Memory optimized using
 * prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <set>

#include <hip/hip_runtime.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

const char *sSDKname = "conjugateGradientMultiDeviceCG";

#define ENABLE_CPU_DEBUG_CODE 0
#define THREADS_PER_BLOCK 512

__device__ double grid_dot_result = 0.0;

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz) {
  I[0] = 0, J[0] = 0, J[1] = 1;
  val[0] = (float)rand() / RAND_MAX + 10.0f;
  val[1] = (float)rand() / RAND_MAX;
  int start;

  for (int i = 1; i < N; i++) {
    if (i > 1) {
      I[i] = I[i - 1] + 3;
    } else {
      I[1] = 2;
    }

    start = (i - 1) * 3 + 2;
    J[start] = i - 1;
    J[start + 1] = i;

    if (i < N - 1) {
      J[start + 2] = i + 1;
    }

    val[start] = val[start - 1];
    val[start + 1] = (float)rand() / RAND_MAX + 10.0f;

    if (i < N - 1) {
      val[start + 2] = (float)rand() / RAND_MAX;
    }
  }

  I[N] = nz;
}

// I - contains location of the given non-zero element in the row of the matrix
// J - contains location of the given non-zero element in the column of the
// matrix val - contains values of the given non-zero elements of the matrix
// inputVecX - input vector to be multiplied
// outputVecY - resultant vector
void cpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha,
             float *inputVecX, float *outputVecY) {
  for (int i = 0; i < num_rows; i++) {
    int num_elems_this_row = I[i + 1] - I[i];

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      output += alpha * val[I[i] + j] * inputVecX[J[I[i] + j]];
    }
    outputVecY[i] = output;
  }

  return;
}

double dotProduct(float *vecA, float *vecB, int size) {
  double result = 0.0;

  for (int i = 0; i < size; i++) {
    result = result + (vecA[i] * vecB[i]);
  }

  return result;
}

void scaleVector(float *vec, float alpha, int size) {
  for (int i = 0; i < size; i++) {
    vec[i] = alpha * vec[i];
  }
}

void saxpy(float *x, float *y, float a, int size) {
  for (int i = 0; i < size; i++) {
    y[i] = a * x[i] + y[i];
  }
}

void cpuConjugateGrad(int *I, int *J, float *val, float *x, float *Ax, float *p,
                      float *r, int nnz, int N, float tol) {
  int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, b, a, na;

  cpuSpMV(I, J, val, nnz, N, alpha, x, Ax);
  saxpy(Ax, r, alpham1, N);

  float r1 = dotProduct(r, r, N);

  int k = 1;

  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;
      scaleVector(p, b, N);

      saxpy(r, p, alpha, N);
    } else {
      for (int i = 0; i < N; i++) p[i] = r[i];
    }

    cpuSpMV(I, J, val, nnz, N, alpha, p, Ax);

    float dot = dotProduct(p, Ax, N);
    a = r1 / dot;

    saxpy(p, x, a, N);
    na = -a;
    saxpy(Ax, r, na, N);

    r0 = r1;
    r1 = dotProduct(r, r, N);

    printf("\nCPU code iteration = %3d, residual = %e\n", k, sqrt(r1));
    k++;
  }
}

__device__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows,
                        float alpha, float *inputVecX, float *outputVecY,
                        cg::thread_block &cta,
                        const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < num_rows; i += multi_grid.size()) {
    int row_elem = I[i];
    int next_row_elem = I[i + 1];
    int num_elems_this_row = next_row_elem - row_elem;

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      output += alpha * val[row_elem + j] * inputVecX[J[row_elem + j]];
    }

    outputVecY[i] = output;
  }
}

__device__ void gpuSaxpy(float *x, float *y, float a, int size,
                         const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    y[i] = a * x[i] + y[i];
  }
}

__device__ void gpuDotProduct(float *vecA, float *vecB, int size,
                              const cg::thread_block &cta,
                              const cg::multi_grid_group &multi_grid) {
  __shared__ double tmp[THREADS_PER_BLOCK];

  double temp_sum = 0.0;
  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    temp_sum += (double)(vecA[i] * vecB[i]);
  }
  tmp[cta.thread_rank()] = temp_sum;

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  double beta = temp_sum;
  double temp;

  for (int i = tile32.size() / 2; i > 0; i >>= 1) {
    if (tile32.thread_rank() < i) {
      temp = tmp[cta.thread_rank() + i];
      beta += temp;
      tmp[cta.thread_rank()] = beta;
    }
    cg::sync(tile32);
  }
  cg::sync(cta);

  if (cta.thread_rank() == 0) {
    beta = 0.0;
    for (int i = 0; i < cta.size(); i += tile32.size()) {
      beta += tmp[i];
    }
    atomicAdd(&grid_dot_result, beta);
  }
}

__device__ void gpuCopyVector(float *srcA, float *destB, int size,
                              const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    destB[i] = srcA[i];
  }
}

__device__ void gpuScaleVector(float *vec, float alpha, int size,
                               const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    vec[i] = alpha * vec[i];
  }
}

__device__ void setDotResultToZero(double *dot_result) {
  unsigned long long int *address_as_ull = (unsigned long long int *)dot_result;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS_system(address_as_ull, assumed, 0);

  } while (assumed != old);
}

extern "C" __global__ void multiGpuConjugateGradient(
    int *I, int *J, float *val, float *x, float *Ax, float *p, float *r,
    double *dot_result, int nnz, int N, float tol) {
  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();
  cg::multi_grid_group multi_grid = cg::this_multi_grid();

  const int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, r1, b, a, na;

  for (int i = multi_grid.thread_rank(); i < N; i += multi_grid.size()) {
    r[i] = 1.0;
    x[i] = 0.0;
  }

  cg::sync(grid);

  gpuSpMV(I, J, val, nnz, N, alpha, x, Ax, cta, multi_grid);

  cg::sync(grid);

  gpuSaxpy(Ax, r, alpham1, N, multi_grid);

  cg::sync(grid);

  gpuDotProduct(r, r, N, cta, multi_grid);

  cg::sync(grid);

  if (grid.thread_rank() == 0) {
    atomicAdd_system(dot_result, grid_dot_result);
    grid_dot_result = 0.0;
  }
  cg::sync(multi_grid);

  r1 = *dot_result;

  int k = 1;
  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;

      gpuScaleVector(p, b, N, multi_grid);
      cg::sync(grid);
      gpuSaxpy(r, p, alpha, N, multi_grid);
    } else {
      gpuCopyVector(r, p, N, multi_grid);
    }

    cg::sync(multi_grid);

    gpuSpMV(I, J, val, nnz, N, alpha, p, Ax, cta, multi_grid);

    if (multi_grid.thread_rank() == 0) {
      setDotResultToZero(dot_result);
    }
    cg::sync(multi_grid);

    gpuDotProduct(p, Ax, N, cta, multi_grid);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
      atomicAdd_system(dot_result, grid_dot_result);
      grid_dot_result = 0.0;
    }
    cg::sync(multi_grid);

    a = r1 / *dot_result;

    gpuSaxpy(p, x, a, N, multi_grid);

    na = -a;

    gpuSaxpy(Ax, r, na, N, multi_grid);

    r0 = r1;

    cg::sync(multi_grid);
    if (multi_grid.thread_rank() == 0) {
      setDotResultToZero(dot_result);
    }

    cg::sync(multi_grid);

    gpuDotProduct(r, r, N, cta, multi_grid);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
      atomicAdd_system(dot_result, grid_dot_result);
      grid_dot_result = 0.0;
    }
    cg::sync(multi_grid);

    r1 = *dot_result;
    k++;
  }
}

void getIdenticalGPUs(int num_of_gpus, std::set<int> &identicalGPUs) {
  int *major_minor = (int *)malloc(sizeof(int) * num_of_gpus * 2);
  int foundIdenticalGPUs = 0;

  for (int i = 0; i < num_of_gpus; i++) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, i));
    major_minor[i * 2] = deviceProp.major;
    major_minor[i * 2 + 1] = deviceProp.minor;
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i,
           deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  int maxMajorMinor[2] = {0, 0};

  for (int i = 0; i < num_of_gpus; i++) {
    for (int j = i + 1; j < num_of_gpus; j++) {
      if ((major_minor[i * 2] == major_minor[j * 2]) &&
          (major_minor[i * 2 + 1] == major_minor[j * 2 + 1])) {
        identicalGPUs.insert(i);
        identicalGPUs.insert(j);
        foundIdenticalGPUs = 1;
        if (maxMajorMinor[0] < major_minor[i * 2] &&
            maxMajorMinor[1] < major_minor[i * 2 + 1]) {
          maxMajorMinor[0] = major_minor[i * 2];
          maxMajorMinor[1] = major_minor[i * 2 + 1];
        }
      }
    }
  }

  free(major_minor);
  if (!foundIdenticalGPUs) {
    printf(
        "No Two or more GPUs with same architecture found\nWaiving the "
        "sample\n");
    exit(EXIT_WAIVED);
  }

  std::set<int>::iterator it = identicalGPUs.begin();

  // Iterate over all the identical GPUs found
  while (it != identicalGPUs.end()) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, *it));
    // Remove all the GPUs which are less than the best arch available
    if (deviceProp.major != maxMajorMinor[0] &&
        deviceProp.minor != maxMajorMinor[1]) {
      identicalGPUs.erase(it);
    }
    if (!deviceProp.cooperativeMultiDeviceLaunch ||
        !deviceProp.concurrentManagedAccess) {
      identicalGPUs.erase(it);
    }
    it++;
  }

  return;
}

int main(int argc, char **argv) {
  int N = 0, nz = 0, *I = NULL, *J = NULL;
  float *val = NULL;
  const float tol = 1e-5f;
  float *x;
  float rhs = 1.0;
  float r1;
  float *r, *p, *Ax;

  printf("Starting [%s]...\n", sSDKname);

  int num_of_gpus = 0;
  checkCudaErrors(hipGetDeviceCount(&num_of_gpus));

  if (num_of_gpus <= 1) {
    printf("No. of GPU on node %d\n", num_of_gpus);
    printf("Minimum Two or more GPUs are required to run this sample code\n");
    exit(EXIT_WAIVED);
  }

  std::set<int> identicalGPUs;
  getIdenticalGPUs(num_of_gpus, identicalGPUs);

  if (identicalGPUs.size() <= 1) {
    printf(
        "No Two or more GPUs with same architecture capable of "
        "cooperativeMultiDeviceLaunch & concurrentManagedAccess found. \nWaiving the sample\n");
    exit(EXIT_WAIVED);
  }

  std::set<int>::iterator deviceId = identicalGPUs.begin();

  // We use only 2 GPUs as for input size of N = 10485760*2 two GPUs are enough.
  while (identicalGPUs.size() > 2) {
    identicalGPUs.erase(deviceId);
    deviceId++;
  }
  /* Generate a random tridiagonal symmetric matrix in CSR format */
  N = 10485760 * 2;
  nz = (N - 2) * 3 + 4;

  checkCudaErrors(hipMallocManaged((void **)&I, sizeof(int) * (N + 1)));
  checkCudaErrors(hipMallocManaged((void **)&J, sizeof(int) * nz));
  checkCudaErrors(hipMallocManaged((void **)&val, sizeof(float) * nz));

  float *val_cpu = (float *)malloc(sizeof(float) * nz);

  genTridiag(I, J, val_cpu, N, nz);

  memcpy(val, val_cpu, sizeof(float) * nz);
  checkCudaErrors(
      hipMemAdvise(I, sizeof(int) * (N + 1), hipMemAdviseSetReadMostly, 0));
  checkCudaErrors(
      hipMemAdvise(J, sizeof(int) * nz, hipMemAdviseSetReadMostly, 0));
  checkCudaErrors(
      hipMemAdvise(val, sizeof(float) * nz, hipMemAdviseSetReadMostly, 0));

  checkCudaErrors(hipMallocManaged((void **)&x, sizeof(float) * N));

  double *dot_result;
  checkCudaErrors(hipMallocManaged((void **)&dot_result, sizeof(double)));

  checkCudaErrors(hipMemset(dot_result, 0.0, sizeof(double)));

  // temp memory for ConjugateGradient
  checkCudaErrors(hipMallocManaged((void **)&r, N * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&p, N * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&Ax, N * sizeof(float)));

  std::cout << "\nRunning on GPUs = " << identicalGPUs.size() << std::endl;
  hipStream_t *nStreams =
      (hipStream_t *)malloc(sizeof(hipStream_t) * identicalGPUs.size());

  void *kernelArgs[] = {
      (void *)&I,  (void *)&J, (void *)&val, (void *)&x,
      (void *)&Ax, (void *)&p, (void *)&r,   (void *)&dot_result,
      (void *)&nz, (void *)&N, (void *)&tol,
  };

  int sMemSize = sizeof(double) * THREADS_PER_BLOCK;
  int numBlocksPerSm = 0;
  int numThreads = THREADS_PER_BLOCK;

  deviceId = identicalGPUs.begin();
  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipSetDevice(*deviceId));
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, *deviceId));

  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &numBlocksPerSm, multiGpuConjugateGradient, numThreads, sMemSize));

  int numSms = deviceProp.multiProcessorCount;
  dim3 dimGrid(numSms * numBlocksPerSm, 1, 1),
      dimBlock(THREADS_PER_BLOCK, 1, 1);

  int device_count = 0;

  int totalThreadsPerGPU = numSms * numBlocksPerSm * THREADS_PER_BLOCK;

  while (deviceId != identicalGPUs.end()) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, *deviceId));
    checkCudaErrors(hipStreamCreate(&nStreams[device_count]));

    if (deviceProp.concurrentManagedAccess) {
      int perGPUIter = N / (totalThreadsPerGPU * identicalGPUs.size());
      int offset_Ax = device_count * totalThreadsPerGPU;
      int offset_r = device_count * totalThreadsPerGPU;
      int offset_p = device_count * totalThreadsPerGPU;
      int offset_x = device_count * totalThreadsPerGPU;

      checkCudaErrors(hipMemPrefetchAsync(I, sizeof(int) * N, *deviceId,
                                           nStreams[device_count]));
      checkCudaErrors(hipMemPrefetchAsync(val, sizeof(float) * nz, *deviceId,
                                           nStreams[device_count]));
      checkCudaErrors(hipMemPrefetchAsync(J, sizeof(float) * nz, *deviceId,
                                           nStreams[device_count]));

      if (offset_Ax <= N) {
        for (int i = 0; i < perGPUIter; i++) {
          hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetPreferredLocation, *deviceId);
          hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetPreferredLocation, *deviceId);
          hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetPreferredLocation, *deviceId);
          hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetPreferredLocation, *deviceId);

          hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetAccessedBy, *deviceId);
          hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetAccessedBy, *deviceId);
          hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetAccessedBy, *deviceId);
          hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                        hipMemAdviseSetAccessedBy, *deviceId);

          offset_Ax += totalThreadsPerGPU * identicalGPUs.size();
          offset_r += totalThreadsPerGPU * identicalGPUs.size();
          offset_p += totalThreadsPerGPU * identicalGPUs.size();
          offset_x += totalThreadsPerGPU * identicalGPUs.size();

          if (offset_Ax >= N) {
            break;
          }
        }
      }
    }
    device_count++;
    deviceId++;
  }

#if ENABLE_CPU_DEBUG_CODE
  float *Ax_cpu = (float *)malloc(sizeof(float) * N);
  float *r_cpu = (float *)malloc(sizeof(float) * N);
  float *p_cpu = (float *)malloc(sizeof(float) * N);
  float *x_cpu = (float *)malloc(sizeof(float) * N);

  for (int i = 0; i < N; i++) {
    r_cpu[i] = 1.0;
    Ax_cpu[i] = x_cpu[i] = 0.0;
  }
#endif

  printf("Total threads per GPU = %d numBlocksPerSm  = %d\n",
         numSms * numBlocksPerSm * THREADS_PER_BLOCK, numBlocksPerSm);
  hipLaunchParams *launchParamsList = (hipLaunchParams *)malloc(
      sizeof(hipLaunchParams) * identicalGPUs.size());
  for (int i = 0; i < identicalGPUs.size(); i++) {
    launchParamsList[i].func = (void *)multiGpuConjugateGradient;
    launchParamsList[i].gridDim = dimGrid;
    launchParamsList[i].blockDim = dimBlock;
    launchParamsList[i].sharedMem = sMemSize;
    launchParamsList[i].stream = nStreams[i];
    launchParamsList[i].args = kernelArgs;
  }

  printf("Launching kernel\n");
  checkCudaErrors(hipLaunchCooperativeKernelMultiDevice(
      launchParamsList, identicalGPUs.size(),
      hipCooperativeLaunchMultiDeviceNoPreSync |
          hipCooperativeLaunchMultiDeviceNoPostSync));

  if (deviceProp.concurrentManagedAccess) {
    checkCudaErrors(
        hipMemPrefetchAsync(x, sizeof(float) * N, hipCpuDeviceId));
    checkCudaErrors(
        hipMemPrefetchAsync(dot_result, sizeof(double), hipCpuDeviceId));
  }

  deviceId = identicalGPUs.begin();
  device_count = 0;
  while (deviceId != identicalGPUs.end()) {
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipStreamSynchronize(nStreams[device_count++]));
    deviceId++;
  }

  r1 = *dot_result;

  printf("GPU Final, residual = %e \n  ", sqrt(r1));

#if ENABLE_CPU_DEBUG_CODE
  cpuConjugateGrad(I, J, val, x_cpu, Ax_cpu, p_cpu, r_cpu, nz, N, tol);
#endif

  float rsum, diff, err = 0.0;

  for (int i = 0; i < N; i++) {
    rsum = 0.0;

    for (int j = I[i]; j < I[i + 1]; j++) {
      rsum += val_cpu[j] * x[J[j]];
    }

    diff = fabs(rsum - rhs);

    if (diff > err) {
      err = diff;
    }
  }

  checkCudaErrors(hipFree(I));
  checkCudaErrors(hipFree(J));
  checkCudaErrors(hipFree(val));
  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(r));
  checkCudaErrors(hipFree(p));
  checkCudaErrors(hipFree(Ax));
  checkCudaErrors(hipFree(dot_result));
  free(val_cpu);

#if ENABLE_CPU_DEBUG_CODE
  free(Ax_cpu);
  free(r_cpu);
  free(p_cpu);
  free(x_cpu);
#endif

  printf("Test Summary:  Error amount = %f \n", err);
  fprintf(stdout, "&&&& conjugateGradientMultiDeviceCG %s\n",
          (sqrt(r1) < tol) ? "PASSED" : "FAILED");
  exit((sqrt(r1) < tol) ? EXIT_SUCCESS : EXIT_FAILURE);
}
