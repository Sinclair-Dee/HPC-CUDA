#include "hip/hip_runtime.h"
#include "Net.h"

namespace GPU_Scope{

void Convolution::init(int minib, int Inputimage_h, int Inputimage_w, int Inputimage_ch, int W_w_h, int W_ch){
  std::default_random_engine generator;
  std::normal_distribution<float> distribution(0,0.1);

  this->MiniBatch = minib;

  this->W_weight_height = W_w_h;
  this->W_channel = W_ch;//一维表达

  this->Inputimage_width = Inputimage_w;
  this->Inputimage_height = Inputimage_h;
  this->Inputimage_channel = Inputimage_ch;
  this->FM_in_width = Inputimage_ch * Inputimage_w * Inputimage_h;
  this->FM_in_height = minib;

  this->Outputimage_width = (Inputimage_width-W_width_height + 1);
  this->Outputimage_height = (Inputimage_height-W_width_height + 1);
  this->Outputimage_channel = W_channel / Inputimage_channel;
  this->FM_out_weight = Outputimage_channel * Outputimage_height * Outputimage_width;
  this->FM_out_height = minib;

  this->Unroll_FM_in_width  = Outputimage_width * Outputimage_height;
  this->Unroll_FM_in_height = Inputimage_channel * W_width_height * W_width_height;

  //allocate memory
  //template<typename T, typename Alloc = thrust::device_malloc_allocator<T>>
  //void thrust::device_vector< T, Alloc >::resize	(	size_type 	new_size,
  //                                                  const value_type & 	x = value_type())
  // new_size	Number of elements this vector should contain.
  // x	Data with which new elements should be populated.
  this->device_FM_in.resize(Mini_Batch * Inputimage_channel * Inputimage_height * Inputimage_width, 0);
  this->host_FM_in.resize(Mini_Batch * Inputimage_channel * Inputimage_height * Inputimage_width, 0);
  this->device_Unroll_FM_in.resize((Inputimage_channel * W_width_height * W_width_height) * (Outputimage_width *Outputimage_height),0);
  this->device_Unroll_FM_inT.resize((Outputimage_width*Outputimage_height) * (Inputimage_channel*W_width_height*W_width_height),0);
  this->host_Unroll_FM_in.resize((Inputimage_channel*W_width_height*W_width_height) * (Outputimage_width*Outputimage_height),0);
  this->host_W.resize(W_channel * W_width_height * W_width_height, 0.5);
  this->device_W.resize(W_channel * W_width_height * W_width_height, 0.5);
  this->device_WT.resize(W_channel * W_width_height * Outputimage_channel, 0.5);
  for(int i = 0; i < W_channel*W_width_height*W_width_height; i++){this->host_W[i] =  distribution(generator);}
  for(int i = 0; i < W_channel*W_width_height*W_width_height; i++){this->device_W[i] = distribution(generator);}
  this->host_FM_out.resize(MiniBatch * Outputimage_channel * Outputimage_width * Outputimage_height, 0);
  this->device_FM_out.resize(MiniBatch * Outputimage_channel * Outputimage_width * Outputimage_height, 0);
  this->host_Wgrad.resize(Outputimage_channel * Inputimage_channel * W_width_height * W_width_height, 0);
  this->device_Wgrad.resize(Outputimage_channel * Inputimage_channel * W_width_height * W_width_height, 0);
  this->device_Wgrad_Temp.resize(Outputimage_channel * Inputimage_channel * W_width_height * W_width_height, 0);
}

void Convlution::forward_CPU(){
  float *input_pointer = thrust::raw_pointer_cast(host_FM_in,data());
  float *W_pointer = thrust::raw_pointer_cast(host_W.data());
  float *Output_pointer = thrust::raw_pointer_cast(host_FM_out());
  ConvLayerForward(MiniBatch,input_pointer,Inputimage_channel, Inputimage_height,Inputimage_width, W_pointer, W_width_height, Output_pointer, Outputimage_channel);
}

void Convlution::forward_CPU_naive(){
  dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
  int bz = ceil((float)Outputimage_width/TILE_WIDTH) * ceil((float)Outputimage_height/TILE_WIDTH);
  bz = bz == 0 ? 1 : bz
  dim3 nunBlocks(MiniBatch, Outputimage_channel , bz);

  float *input_pointer = thrust::raw_pointer_cast(device_FM_in.data());
  float *W_pointer = thrust::raw_pointer_cast(device_W.data());
  float *Output_pointer = thrust::raw_pointer_cast(device_FM_out.data());

  ConvLayerForwardGPUnaive<<<numBlocks, threadsPerBlock>>>(input_pointer, W_pointer, Output_pointer,
                                  Inputimage_channel, Inputimage_height, Inputimage_width , Outputimage_width, W_width_height, Outputimage_channel);


}

void Convolution::forward_CPU(){
  float *input_pointer = thrust::raw_pointer_cast(host_FM_in.data());
  float *W_pointer = thrust::raw_pointer_cast(host_W.data());
  float *Output_point = thrust::raw_pointer_cast(host_FM_out.data());
  //call function ConvLayerForward in Function.cu
  ConvLayerForward(MiniBatch, input_pointer, Inputimage_channel, Inputimage_height,
                    Inputimage_width, W_pointer, W_width_height, Output_pointer, Outputimage_channel);
}

void Convolution::forward_GPU_naive(){
	dim3 threadsPerBlock(TILE_WIDTH,TILE_WIDTH);
	int bz = ceil((float)Outputimage_width/TILE_WIDTH) * ceil((float)Outputimage_height/TILE_WIDTH);
	if( bz == 0 )
		bz = 1;
	dim3 numBlocks(minibatch, Outputimage_channel , bz);

	float* input_pointer = thrust::raw_pointer_cast( device_FM_in.data() );
	float* W_pointer = thrust::raw_pointer_cast( device_W.data() );
	float* Output_pointer = thrust::raw_pointer_cast( device_FM_out.data() );

	ConvLayerForwardGPUnaive<<<numBlocks,threadsPerBlock>>>(input_pointer, W_pointer, Output_pointer,
			Inputimage_channel, Inputimage_height, Inputimage_width , Outputimage_width, W_width_height, Outputimage_channel);
}

void Convolution::forward_GPU_tiled(){
  dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
  int bz = ceil((float)Outputimage_width / TILE_WIDTH) * ceil((float)Outputimage_height / TILE_height);
  bz = bz == 0 ? 1 :bz;
  dim3 numBlocks(MiniBatch, Outputimage_channel, bz);

  float *input_pointer = thrust::raw_pointer_cast(device_FM_in.data());
  float *W_pointer = thrust::raw_pointer_cast(device_W.data());
  float *Output_point = thrust::raw_pointer_cast(device_FM_out.data());

  ConvLayerForwardGPUtiled<<<mumBlocks, threadsPerBlock>>>(input_pointer, W_pointer, Output_pointer,
			Inputimage_channel, Inputimage_height, Inputimage_width , Outputimage_width, W_width_height, Outputimage_channel);

}

void Convlution::forward_GPU_gemm(){
  //device_W -> (height)Outputimage_channel*(width)Inputimage_channel*W_width_height*W_width_height
  //device_FM_in  -> (height)Inputimage_channel*W_width_height*W_width_height*(width)Outputimage_width*Outputimage_height
  //device_FM_out -> (height)Outputimage_channel*(width)Outputimage_width*Outputimage_height
  float* Output_pointer = thrust::raw_pointer_cast( device_FM_out.data() );
  float* FM_in__pointer = thrust::raw_pointer_cast( device_FM_in.data() );
  float* Unroll_FM_in_pointer = thrust::raw_pointer_cast( device_Unroll_FM_in.data() );

  for(int i = 0; i< MiniBatch; i++){
    int H_out = Inputimage_height - W_h_w + 1;
    int W_out = Inputimage_width - W_h_w + 1;

    int num_Thread = Inputimage_channel * Outputimage_height*Outputimage_width;
    int num_Blocks = ceil((float)num_Thread/1024);

    unroll_Kernel<<<num_blocks, 1024>>>(Inputimage_channel, Inputimage_height,Inputimage_width, W_width_height,
                                        FM_in_pointer, Unroll_FM_in_pointer);

    float *W_pointer = thust::raw_pointer_cast(device_W.data());

    dim3 threadsPerBlock(TILE_WIDTH,TILE_WIDTH)
    dim3 numBlocks(ceil((float)Outputimage_width * Outputimage_height/TILE_WIDTH), ceil((float)Outputimage_channel/TILE_WIDTH));

    //void GEMM(float *W, float* Unroll_FM_in, float* FM_out, int M_height_in, int M_width_N_height_in, int N_width_in, int height_out, int width_out);
    GEMM<<<numBlocks,threadsPerBlock>>>(W_pointer, Unroll_FM_in_pointer, Output_pointer,
           Outputimage_channel,Inputimage_channel*W_width_height*W_width_height, Outputimage_width*Outputimage_height,
           Outputimage_channel, Outputimage_width*Outputimage_height);

    Output_pointer = Output_pointer+(Outputimage_channel*Outputimage_width*Outputimage_height);
    FM_in__pointer = FM_in__pointer + (Inputimage_channel*Inputimage_height*Inputimage_width);
  }
}

void Convlution::backward_GPU_gemm(){


}
}

namespace FPGA_HLS_Scope{//to do
}
