#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp> 
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using namespace std;
using namespace cv;

//声明CUDA纹理
texture <uchar4, hipTextureType2D, hipReadModeNormalizedFloat> refTex1;
texture <uchar4, hipTextureType2D, hipReadModeNormalizedFloat> refTex2;
//声明CUDA数组
hipArray* cuArray1;
hipArray* cuArray2;
//通道数
hipChannelFormatDesc cuDesc = hipCreateChannelDesc<uchar4>();


__global__ void weightAddKerkel(uchar *pDstImgData, int imgHeight, int imgWidth,int channels)
{
    const int tidx=blockDim.x*blockIdx.x+threadIdx.x;
    const int tidy=blockDim.y*blockIdx.y+threadIdx.y;

    if (tidx<imgWidth && tidy<imgHeight)
    {
        float4 lenaBGR,moonBGR;
        //使用tex2D函数采样纹理
        lenaBGR=tex2D(refTex1, tidx, tidy);
        moonBGR=tex2D(refTex2, tidx, tidy);

        int idx=(tidy*imgWidth+tidx)*channels;
        float alpha=0.5;
        pDstImgData[idx+0]=(alpha*lenaBGR.x+(1-alpha)*moonBGR.x)*255;
        pDstImgData[idx+1]=(alpha*lenaBGR.y+(1-alpha)*moonBGR.y)*255;
        pDstImgData[idx+2]=(alpha*lenaBGR.z+(1-alpha)*moonBGR.z)*255;
        pDstImgData[idx+3]=0;
    }
}

int  main()
{
    Mat Lena=imread("lena.jpg");
    Mat moon=imread("moon.jpg");
    cvtColor(Lena, Lena, CV_BGR2BGRA);
    cvtColor(moon, moon, CV_BGR2BGRA);
    int imgWidth=Lena.cols;
    int imgHeight=Lena.rows;
    int channels=Lena.channels();

    //设置纹理属性
    hipError_t t;
    refTex1.addressMode[0] = hipAddressModeClamp;
    refTex1.addressMode[1] = hipAddressModeClamp;
    refTex1.normalized = false;
    refTex1.filterMode = hipFilterModeLinear;
    //绑定cuArray到纹理
    hipMallocArray(&cuArray1, &cuDesc, imgWidth, imgHeight);
    t = hipBindTextureToArray(refTex1, cuArray1);

    refTex2.addressMode[0] = hipAddressModeClamp;
    refTex2.addressMode[1] = hipAddressModeClamp;
    refTex2.normalized = false;
    refTex2.filterMode = hipFilterModeLinear;
     hipMallocArray(&cuArray2, &cuDesc, imgWidth, imgHeight);
    t = hipBindTextureToArray(refTex2, cuArray2);

    //拷贝数据到hipArray
    t=hipMemcpyToArray(cuArray1, 0,0, Lena.data, imgWidth*imgHeight*sizeof(uchar)*channels, hipMemcpyHostToDevice);
    t=hipMemcpyToArray(cuArray2, 0,0, moon.data, imgWidth*imgHeight*sizeof(uchar)*channels, hipMemcpyHostToDevice);

    //输出图像
    Mat dstImg=Mat::zeros(imgHeight, imgWidth, CV_8UC4);
    uchar *pDstImgData=NULL;
    t=hipMalloc(&pDstImgData, imgHeight*imgWidth*sizeof(uchar)*channels);

    //核函数，实现两幅图像加权和
    dim3 block(8,8);
    dim3 grid( (imgWidth+block.x-1)/block.x, (imgHeight+block.y-1)/block.y );
    weightAddKerkel<<<grid, block, 0>>>(pDstImgData, imgHeight, imgWidth, channels);
    hipDeviceSynchronize();

    //从GPU拷贝输出数据到CPU
    t=hipMemcpy(dstImg.data, pDstImgData, imgWidth*imgHeight*sizeof(uchar)*channels, hipMemcpyDeviceToHost);

    //显示
    //namedWindow("show");
    //imshow("show", dstImg);
    //waitKey(0);i
  cv::imwrite("output.jpg",dstImg);
  return 0;
} 
