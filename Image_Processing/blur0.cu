#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

cv::Mat imageInputRGBA;
cv::Mat imageOutputRGBA;

uchar4 *d_inputImageRGBA__;
uchar4 *d_outputImageRGBA__;

float *h_filter__;

size_t numRows() { return imageInputRGBA.rows; }
size_t numCols() { return imageInputRGBA.cols; }

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

void preProcess(uchar4 **h_inputImageRGBA, uchar4 **h_outputImageRGBA,
                uchar4 **d_inputImageRGBA, uchar4 **d_outputImageRGBA,
                unsigned char **d_redBlurred,
                unsigned char **d_greenBlurred,
                unsigned char **d_blueBlurred,
                float **h_filter, int *filterWidth,
                const std::string &filename) {
  //make sure the context initializes ok
  checkCudaErrors(hipFree(0));

  cv::Mat image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
  if (image.empty()) {
    std::cerr << "Couldn't open file: " << filename << std::endl;
    exit(1);
  }

  cv::cvtColor(image, imageInputRGBA, CV_BGR2RGBA);

  //allocate memory for the output
  imageOutputRGBA.create(image.rows, image.cols, CV_8UC4);

  //This shouldn't ever happen given the way the images are created
  //at least based upon my limited understanding of OpenCV, but better to check
  if (!imageInputRGBA.isContinuous() || !imageOutputRGBA.isContinuous()) {
    std::cerr << "Images aren't continuous!! Exiting." << std::endl;
    exit(1);
  }

  *h_inputImageRGBA  = (uchar4 *)imageInputRGBA.ptr<unsigned char>(0);
  *h_outputImageRGBA = (uchar4 *)imageOutputRGBA.ptr<unsigned char>(0);

  const size_t numPixels = numRows() * numCols();
  //allocate memory on the device for both input and output
  checkCudaErrors(hipMalloc(d_inputImageRGBA, sizeof(uchar4) * numPixels));
  checkCudaErrors(hipMalloc(d_outputImageRGBA, sizeof(uchar4) * numPixels));
  checkCudaErrors(hipMemset(*d_outputImageRGBA, 0, numPixels * sizeof(uchar4))); //make sure no memory is left laying around

  //copy input array to the GPU
  checkCudaErrors(hipMemcpy(*d_inputImageRGBA, *h_inputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

  d_inputImageRGBA__  = *d_inputImageRGBA;
  d_outputImageRGBA__ = *d_outputImageRGBA;

  //now create the filter that they will use
  const int blurKernelWidth = 3;
  const float blurKernelSigma = 2.;

  *filterWidth = blurKernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[blurKernelWidth * blurKernelWidth];
  h_filter__ = *h_filter;

  float filterSum = 0.f; //for normalization

  for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
    for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
      float filterValue = expf( -(float)(c * c + r * r) / (2.f * blurKernelSigma * blurKernelSigma));
      (*h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] = filterValue;
      filterSum += filterValue;
    }
  }

  float normalizationFactor = 1.f / filterSum;

  for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
    for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
      (*h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] *= normalizationFactor;
    }
  }

  //blurred
  checkCudaErrors(hipMalloc(d_redBlurred,sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc(d_greenBlurred,sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMalloc(d_blueBlurred,sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_redBlurred,0,sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_greenBlurred,0,sizeof(unsigned char) * numPixels));
  checkCudaErrors(hipMemset(*d_blueBlurred,0,sizeof(unsigned char) * numPixels));//make sure the context initializes ok
  checkCudaErrors(hipFree(0));

}

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  const int absolute_image_position_x = thread_2D_pos.x;
  const int absolute_image_position_y = thread_2D_pos.y;
  if ( absolute_image_position_x >= numCols ||
       absolute_image_position_y >= numRows )
  {
      return;
  }
  float color = 0.0f;
  for(int py=0; py < filterWidth; py++){
    for(int px=0; px < filterWidth; px++){
      int c_x = absolute_image_position_x + px - filterWidth / 2;
      int c_y = absolute_image_position_y + py - filterWidth / 2;
      c_x = min(max(c_x, 0), numCols - 1);
      c_y = min(max(c_y, 0), numRows - 1);
      float filter_value = filter[py*filterWidth + px];
      color += filter_value*static_cast<float>(inputChannel[c_y*numCols + c_x]);
    }
  }
  outputChannel[thread_1D_pos] = color;
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  const int absolute_image_position_x = thread_2D_pos.x;
  const int absolute_image_position_y = thread_2D_pos.y;
  if ( absolute_image_position_x >= numCols ||
       absolute_image_position_y >= numRows )
  {
      return;
  }
  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  checkCudaErrors(hipMalloc(&d_filter, sizeof( float) * filterWidth * filterWidth));
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));

}

void postProcess(const std::string& output_file, uchar4* data_ptr) {
	cv::Mat output(numRows(), numCols(), CV_8UC4, (void*)data_ptr);
  	cv::Mat imageOutputBGR;
  	cv::cvtColor(output, imageOutputBGR, CV_RGBA2BGR);
  	//output the image
  	cv::imwrite(output_file.c_str(), imageOutputBGR);
}

void cleanup(){
  	//cleanup
	hipFree(d_inputImageRGBA__);
  	hipFree(d_outputImageRGBA__);
  	delete[] h_filter__;
}

int main(int argc,char* argv[]){

	//load input file
	std::string input_file = argv[1];
	//define output file
	std::string output_file = argv[2];

	uchar4 *h_inputImageRGBA,  *d_inputImageRGBA;
  	uchar4 *h_outputImageRGBA, *d_outputImageRGBA;
  	unsigned char *d_redBlurred, *d_greenBlurred, *d_blueBlurred;

	float *h_filter;
  	int    filterWidth;

	//load the image and give us our input and output pointers
  	preProcess(&h_inputImageRGBA, &h_outputImageRGBA, &d_inputImageRGBA, &d_outputImageRGBA,
             &d_redBlurred, &d_greenBlurred, &d_blueBlurred,
             &h_filter, &filterWidth, input_file);

	allocateMemoryAndCopyToGPU(numRows(), numCols(), h_filter, filterWidth);

	const dim3 blockSize(16, 16);
  	const dim3 gridSize(numCols()/blockSize.x+1,numRows()/blockSize.y+1);
	
	//Launch a kernel for separating the RGBA image into different color channels
	separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,
                                              numRows(),
                                              numCols(),
                                              d_red,
                                              d_green,
                                              d_blue);

	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	//Call your convolution kernel here 3 times, once for each color channel.
	gaussian_blur<<<gridSize, blockSize>>>(d_red,
                                         d_redBlurred,
                                         numRows(),
                                         numCols(),
                                         d_filter,
                                         filterWidth);
	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());
	gaussian_blur<<<gridSize, blockSize>>>(d_green,
                                         d_greenBlurred,
                                         numRows(),
                                         numCols(),
                                         d_filter,
                                         filterWidth);
        hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());
	gaussian_blur<<<gridSize, blockSize>>>(d_blue,
                                         d_blueBlurred,
                                         numRows(),
                                         numCols(),
                                         d_filter,
                                         filterWidth);
        hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	// Now we recombine your results. We take care of launching this kernel for you.
  	//
  	// NOTE: This kernel launch depends on the gridSize and blockSize variables,
  	// which you must set yourself.
	recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows(),
                                             numCols());
	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	size_t numPixels = numRows()*numCols();
  	//copy the output back to the host
  	checkCudaErrors(hipMemcpy(h_outputImageRGBA, d_outputImageRGBA__, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));

  	postProcess(output_file, h_outputImageRGBA);

	checkCudaErrors(hipFree(d_redBlurred));
  	checkCudaErrors(hipFree(d_greenBlurred));
  	checkCudaErrors(hipFree(d_blueBlurred));

	cleanup();

	return 0;
}

