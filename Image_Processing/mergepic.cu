#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using namespace std;
using namespace cv;

//声明CUDA纹理:texture<type,dimension,readtype> texreference;
texture <uchar4,hipTextureType2D,hipReadModeNormalizedFloat> refTex1;
texture <uchar4,hipTextureType2D,hipReadModeNormalizedFloat> refTex2;

////声明CUDA数组
hipArray* cuArray1;
hipArray* cuArray2;

//通道数
hipChannelFormatDesc cuDesc = hipCreateChannelDesc<uchar4>();

__global__ 
void weightAddKernel(uchar *pDstImgData, int imgHeight, int imgWidth,int channels){
  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  const int tidy = blockDim.y * blockIdx.y + threadIdx.y;

  if(tidx<imgWidth && tidy<imgHeight){
    float4 lenaBGR, moonBGR;
    //使用tex2D函数采样纹理
    lenaBGR = tex2D(refTex1,tidx,tidy);
    moonBGR = tex2D(refTex2,tidx,tidy);

    int idx = (tidy * imgWidth + tidx) * channels;
    float alpha = 0.5;
    pDstImgData[idx+0]=(alpha*lenaBGR.x+(1-alpha)*moonBGR.x)*255;
    pDstImgData[idx+1]=(alpha*lenaBGR.y+(1-alpha)*moonBGR.y)*255;
    pDstImgData[idx+2]=(alpha*lenaBGR.z+(1-alpha)*moonBGR.z)*255;
    pDstImgData[idx+3]=0;
  }
}

int main(){
  cv::Mat lena=imread("lena.jpg");
  cv::Mat moon=imread("moon.jpg");

  cvtColor(lena,lena,CV_BGR2BGRA);
  cvtColor(moon,moon,CV_BGR2BGRA);
  int imgWidth = lena.cols;
  int imgHeight = lena.rows;
  int channels = lena.channels();

  //设置refTex1纹理属性
  hipError_t t;
  refTex1.addressMode[0] = hipAddressModeClamp;
  refTex1.addressMode[1] = hipAddressModeClamp;
  refTex1.normalized = false;
  refTex1.filterMode = hipFilterModeLinear;
  //绑定cuArray到纹理refTex1
  hipMallocArray(&cuArray1,&cuDesc,imgWidth,imgHeight);
  t = hipBindTextureToArray(refTex1,cuArray1);
  //设置refTex2纹理属性
  refTex2.addressMode[0] = hipAddressModeClamp;
  refTex2.addressMode[1] = hipAddressModeClamp;
  refTex2.normalized = false;
  refTex2.filterMode = hipFilterModeLinear;
  //绑定cuArray到纹理refTex2
  hipMallocArray(&cuArray2, &cuDesc, imgWidth, imgHeight);
  t = hipBindTextureToArray(refTex2,cuArray2);
  
  //拷贝数据到hipArray
  t = hipMemcpyToArray(cuArray1,0,0,lena.data,imgWidth*imgHeight*sizeof(uchar)*channels, hipMemcpyHostToDevice);
  t = hipMemcpyToArray(cuArray2,0,0,moon.data,imgWidth*imgHeight*sizeof(uchar)*channels, hipMemcpyHostToDevice);
  
  //输出图像
  Mat dstImg = Mat::zeros(imgHeight, imgWidth, CV_8UC4);
  uchar *pDstImgData = NULL;
  t = hipMalloc((uchar **)&pDstImgData, imgHeight*imgWidth*sizeof(uchar)*channels);

  //invoke the kernel
  dim3 block(16,16);
  dim3 grid((imgWidth+block.x-1)/block.x, (imgHeight+block.y-1)/block.y);
  weightAddKernel<<<grid, block, 0>>>(pDstImgData, imgHeight, imgWidth, channels);
  t = hipDeviceSynchronize();

  //从GPU拷贝输出到CPU
  t=hipMemcpy(dstImg.data, pDstImgData, imgWidth*imgHeight*sizeof(uchar)*channels, hipMemcpyDeviceToHost);

  //显示
//  namedWindow("show");
//  imshow("show",dstImg);
//  waitKey();
  
  //存储
  cv::imwrite("hill_merge.jpg",dstImg);
    
  //unbind and free memory
  hipUnbindTexture(refTex1);
  hipUnbindTexture(refTex2);
  hipFreeArray(cuArray1);
  hipFreeArray(cuArray2);

  return 0;
}
